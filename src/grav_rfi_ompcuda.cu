#include "hip/hip_runtime.h"
# include "grav_rfi_ompcuda.h"
# include <stdlib.h>
# include <stdio.h>
# include <string.h>
# include <math.h>
# include <omp.h>
# include <hip/hip_runtime.h>
# include <hip/hip_runtime.h>
# include <>

struct Vz_struct
{
	double* Vz, * x_obs, * y_obs;
	double* m1_x, * m1_y, * m1_z;
};
struct input_struct
{
	double* h_Vz, * h_x_obs, * h_y_obs, * h_m1_x, * h_m1_y, * h_m1_z;
	double* d_Vz, * d_x_obs, * d_y_obs, * d_m1_x, * d_m1_y, * d_m1_z;
	hipStream_t stream;
};
struct Vz_mat_mc_struct
{
	double* d_Vz_mat_mc;
	int nBlocks;
	hipStream_t stream;
};
struct rfi_struct
{
	int localNum, nBlocks, base, nBlocks_Am, nBlocks_Gm;
	double* d_local_Wm, * d_local_Wv, * d_local_W, * d_local_g, * d_local_g0, * d_local_p, * d_local_p0, * d_local_q1, * d_local_q2;
	double* d_local_m_temp, * d_local_m_real, * d_local_d_fit1, * d_local_d_fit2, * d_local_d_fit_temp1, * d_local_d_fit_temp2, * d_d_fit_temp;
	double* h_local_g, * h_local_g0, * h_local_q, * h_local_m_real, * h_local_d_fit, * h_local_d_fit_temp;
	hipStream_t stream;
};

void xy_cmp(int point_count, int lx, int ly, double* x, double* y, double* xmin, double* xmax, double* dx, double* ymin, double* ymax, double* dy);

double* rwt_foc_inv(int deviceCount, int h_point_count, int h_prism_count, int h_lx, int h_ly, int h_lz, int h_kmax,
	double h_z_obs, double h_m_min, double h_m_max, double h_epsilon, double h_lambda, double h_sigma, struct Vz_struct VzX, int Max_GPU_Number, int nThreadPerBlock,double wn);
__global__ void Vz_mat_mc_sln(double* Vz_mat_mc, double* x_obs, double* y_obs, double* m1_x, double* m1_y, double* m1_z, double z_obs, int lx, int lz, int point_count, int prism_count);
__global__ void W_init_sln(double* Wm, double* Wv, double* W, double* m_temp, double* m_real, double* Vz_mat_mc, double sigma, int localNum, int base, int point_count, int lx, double wn);
__global__ void g0_sln(double* g, double* Vz_mat_mc, double* W, double* Vz, int localNum, int base, int point_count, int lx);
__global__ void A_mult_v_col_sln(double* q, double* Vz_mat_mc, double* W, double* vector, int localNum, int base, int point_count, int lx, int nThreadPerBlock);
__global__ void A_mult_v_sum_sln(double* q2, double* q1, double* vector, double lambda, int localNum, int base, int point_count, int prism_count, int nBlocks);
__global__ void m_sln(double* m_temp, double* m_real, double* p, double* W, double alpha, double m_min, double m_max, int localNum);
__global__ void G_mult_m_col_sln(double* d_fit, double* Vz_mat_mc, double* m_temp, int localNum, int base, int point_count, int lx, int nThreadPerBlock);
__global__ void G_mult_m_sum_sln(double* d_fit2, double* d_fit1, int localNum, int point_count, int nBlocks);
__global__ void p_sln(double* p, double* g, double* p0, double beta, int localNum);
__global__ void update_sln(double* p0, double* g0, double* W, double* m_temp, double* p, double* g, double* m_real, double* Wm, double* Wv, int localNum);
__global__ void g_sln(double* g, double* Vz_mat_mc, double* W, double* Vz, double* d_fit_temp, double lambda, int localNum, int base, int point_count, int lx);
double beta_sln(double* g, double* g0, int prism_count);
double vector_dot_product(double* a, double* b, int count);

int CheckCount()
{
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	return deviceCount;
}

double* foo(int h_point_count, int h_prism_count, int h_lx, int h_ly, int h_lz, int h_kmax, int Max_GPU_Number, int nThreadPerBlock,  double h_z_obs, double h_dz, double h_zmax,
	double h_m_min, double h_m_max, double h_epsilon, double h_lambda, double h_sigma,double wn, double *zc, double *thick, double* Vz, double* x, double* y)
{
	struct Vz_struct VzX;
	double xmin, xmax, ymin, ymax, dx, dy;
	double* h_m_result, * m1_x, * m1_y, * m1_z, * x_obs, * y_obs;
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount > Max_GPU_Number)
	{
		deviceCount = Max_GPU_Number;
	}
	xy_cmp(h_point_count, h_lx, h_ly, x, y, &xmin, &xmax, &dx, &ymin, &ymax, &dy);

	x_obs = (double*)malloc(h_lx * sizeof(double));
	y_obs = (double*)malloc(h_ly * sizeof(double));
	for (int xi = 0; xi < h_lx; xi++)
	{
		*(x_obs + xi) = x[xi];
	}
	for (int yi = 0; yi < h_ly; yi++)
	{
		*(y_obs + yi) = y[yi * h_lx];
	}

	m1_x = (double*)malloc(2 * sizeof(double)); *(m1_x + 0) = xmin - 0.5 * dx; *(m1_x + 1) = xmin + 0.5 * dx;
	m1_y = (double*)malloc(2 * sizeof(double)); *(m1_y + 0) = ymin - 0.5 * dy; *(m1_y + 1) = ymin + 0.5 * dy;
	m1_z = (double*)malloc(2 * h_lz * sizeof(double));
	for (int zi = 0; zi < h_lz; zi++)
	{
		*(m1_z + zi) = zc[zi] - 0.5 * thick[zi];
		*(m1_z + zi + h_lz) = zc[zi] + 0.5 * thick[zi];
	}
	VzX.Vz = Vz;
	VzX.x_obs = x_obs;
	VzX.y_obs = y_obs;
	VzX.m1_x = m1_x;
	VzX.m1_y = m1_y;
	VzX.m1_z = m1_z;

	h_m_result = rwt_foc_inv(deviceCount, h_point_count, h_prism_count, h_lx, h_ly, h_lz, h_kmax, h_z_obs, h_m_min, h_m_max, h_epsilon, h_lambda, h_sigma, VzX, Max_GPU_Number, nThreadPerBlock,wn);

	hipDeviceReset();
	return h_m_result;
}

void xy_cmp(int point_count, int lx, int ly, double* x, double* y, double* xmin, double* xmax, double* dx, double* ymin, double* ymax, double* dy)
{
	*xmin = x[0]; *xmax = x[0];
	*ymin = y[0]; *ymax = y[0];
	for (int ni = 1; ni < point_count; ni++)
	{
		*xmin = *xmin < x[ni] ? *xmin : x[ni];
		*xmax = *xmax > x[ni] ? *xmax : x[ni];
		*ymin = *ymin < y[ni] ? *ymin : y[ni];
		*ymax = *ymax > y[ni] ? *ymax : y[ni];
	}
	*dx = (*xmax - *xmin) / (lx - 1);
	*dy = (*ymax - *ymin) / (ly - 1);
}

double* rwt_foc_inv(int deviceCount, int h_point_count, int h_prism_count, int h_lx, int h_ly, int h_lz, int h_kmax,
	double h_z_obs, double h_m_min, double h_m_max, double h_epsilon, double h_lambda, double h_sigma, struct Vz_struct VzX, int Max_GPU_Number, int nThreadPerBlock,double wn)
{
	/*struct input_struct i_struct[Max_GPU_Number_list];
	struct Vz_mat_mc_struct v_struct[Max_GPU_Number_list];
	struct rfi_struct r_struct[Max_GPU_Number_list];*/
	struct input_struct *i_struct = new struct input_struct[Max_GPU_Number];
	struct Vz_mat_mc_struct *v_struct = new struct Vz_mat_mc_struct[Max_GPU_Number];
	struct rfi_struct *r_struct = new struct rfi_struct[Max_GPU_Number];

	int k = 0;
	double alpha, beta, rms, h_d_square, h_phi_m;
	double* h_data_misfit = (double*)malloc(h_point_count * sizeof(double));
	double* h_data_fitting = (double*)malloc(h_point_count * sizeof(double));
	double* h_g = (double*)malloc(h_prism_count * sizeof(double));
	double* h_g0 = (double*)malloc(h_prism_count * sizeof(double));
	double* h_q = (double*)malloc((h_prism_count + h_point_count) * sizeof(double));
	double* h_d_fit_temp = (double*)malloc((h_prism_count + h_point_count) * sizeof(double));
	double* inv_result = (double*)malloc(h_prism_count * sizeof(double));

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		hipStreamCreate(&i_struct[i].stream);
		hipMalloc((void**)&i_struct[i].d_Vz, h_point_count * sizeof(double));
		hipMalloc((void**)&i_struct[i].d_x_obs, h_lx * sizeof(double));
		hipMalloc((void**)&i_struct[i].d_y_obs, h_ly * sizeof(double));
		hipMalloc((void**)&i_struct[i].d_m1_x, 2 * sizeof(double));
		hipMalloc((void**)&i_struct[i].d_m1_y, 2 * sizeof(double));
		hipMalloc((void**)&i_struct[i].d_m1_z, 2 * h_lz * sizeof(double));

		hipHostMalloc((void**)&i_struct[i].h_Vz, h_point_count * sizeof(double));
		hipHostMalloc((void**)&i_struct[i].h_x_obs, h_lx * sizeof(double));
		hipHostMalloc((void**)&i_struct[i].h_y_obs, h_ly * sizeof(double));
		hipHostMalloc((void**)&i_struct[i].h_m1_x, 2 * sizeof(double));
		hipHostMalloc((void**)&i_struct[i].h_m1_y, 2 * sizeof(double));
		hipHostMalloc((void**)&i_struct[i].h_m1_z, 2 * h_lz * sizeof(double));

		memcpy(i_struct[i].h_Vz, VzX.Vz, h_point_count * sizeof(double));
		memcpy(i_struct[i].h_x_obs, VzX.x_obs, h_lx * sizeof(double));
		memcpy(i_struct[i].h_y_obs, VzX.y_obs, h_ly * sizeof(double));
		memcpy(i_struct[i].h_m1_x, VzX.m1_x, 2 * sizeof(double));
		memcpy(i_struct[i].h_m1_y, VzX.m1_y, 2 * sizeof(double));
		memcpy(i_struct[i].h_m1_z, VzX.m1_z, 2 * h_lz * sizeof(double));
	}

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		hipMemcpyAsync(i_struct[i].d_Vz, i_struct[i].h_Vz, h_point_count * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);
		hipMemcpyAsync(i_struct[i].d_x_obs, i_struct[i].h_x_obs, h_lx * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);
		hipMemcpyAsync(i_struct[i].d_y_obs, i_struct[i].h_y_obs, h_ly * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);
		hipMemcpyAsync(i_struct[i].d_m1_x, i_struct[i].h_m1_x, 2 * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);
		hipMemcpyAsync(i_struct[i].d_m1_y, i_struct[i].h_m1_y, 2 * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);
		hipMemcpyAsync(i_struct[i].d_m1_z, i_struct[i].h_m1_z, 2 * h_lz * sizeof(double), hipMemcpyHostToDevice, i_struct[i].stream);

		hipStreamSynchronize(i_struct[i].stream);
	}

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		hipStreamDestroy(i_struct[i].stream);
		hipStreamCreate(&v_struct[i].stream);
		hipMalloc((void**)&v_struct[i].d_Vz_mat_mc, h_prism_count * sizeof(double));

		v_struct[i].nBlocks = h_prism_count / nThreadPerBlock + ((h_prism_count % nThreadPerBlock) ? 1 : 0);
		Vz_mat_mc_sln << <v_struct[i].nBlocks, nThreadPerBlock, 0, v_struct[i].stream >> > (v_struct[i].d_Vz_mat_mc, i_struct[i].d_x_obs, i_struct[i].d_y_obs,
			i_struct[i].d_m1_x, i_struct[i].d_m1_y, i_struct[i].d_m1_z, h_z_obs, h_lx, h_lz, h_point_count, h_prism_count);
		hipStreamSynchronize(v_struct[i].stream);
	}
	for (int i = 0; i < deviceCount; i++)
	{
		r_struct[i].localNum = h_prism_count / deviceCount;
	}
	for (int i = 0; i < h_prism_count % deviceCount; i++)
	{
		r_struct[i].localNum++;
	}

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		hipStreamDestroy(v_struct[i].stream);
		hipStreamCreate(&r_struct[i].stream);

		r_struct[i].nBlocks = r_struct[i].localNum / nThreadPerBlock + ((r_struct[i].localNum % nThreadPerBlock) ? 1 : 0);
		r_struct[i].nBlocks_Am = (h_prism_count + h_point_count) / nThreadPerBlock + (((h_prism_count + h_point_count) % nThreadPerBlock) ? 1 : 0);
		r_struct[i].nBlocks_Gm = h_point_count / nThreadPerBlock + ((h_point_count % nThreadPerBlock) ? 1 : 0);
		r_struct[i].base = 0;
		for (int j = 0; j < i; j++)
		{
			r_struct[i].base += r_struct[j].localNum;
		}
		hipMalloc((void**)&r_struct[i].d_local_Wm, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_Wv, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_W, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_g, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_g0, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_p, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_p0, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_q1, r_struct[i].nBlocks * h_point_count * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_q2, (h_prism_count + h_point_count) * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_m_temp, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_m_real, r_struct[i].localNum * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_d_fit1, r_struct[i].nBlocks * h_point_count * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_d_fit2, h_point_count * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_d_fit_temp1, r_struct[i].nBlocks * h_point_count * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_local_d_fit_temp2, (h_prism_count + h_point_count) * sizeof(double));
		hipMalloc((void**)&r_struct[i].d_d_fit_temp, (h_prism_count + h_point_count) * sizeof(double));

		hipHostMalloc((void**)&r_struct[i].h_local_g, r_struct[i].localNum * sizeof(double));
		hipHostMalloc((void**)&r_struct[i].h_local_g0, r_struct[i].localNum * sizeof(double));
		hipHostMalloc((void**)&r_struct[i].h_local_q, (h_prism_count + h_point_count) * sizeof(double));
		hipHostMalloc((void**)&r_struct[i].h_local_m_real, r_struct[i].localNum * sizeof(double));
		hipHostMalloc((void**)&r_struct[i].h_local_d_fit, h_point_count * sizeof(double));
		hipHostMalloc((void**)&r_struct[i].h_local_d_fit_temp, (h_prism_count + h_point_count) * sizeof(double));

		W_init_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_Wm, r_struct[i].d_local_Wv, r_struct[i].d_local_W,
			r_struct[i].d_local_m_temp, r_struct[i].d_local_m_real, v_struct[i].d_Vz_mat_mc, h_sigma, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx,wn);
		g0_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_g, v_struct[i].d_Vz_mat_mc, r_struct[i].d_local_W,
			i_struct[i].d_Vz, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx);
		hipMemcpyAsync(r_struct[i].h_local_g, r_struct[i].d_local_g, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
		hipStreamSynchronize(r_struct[i].stream);
	}

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		for (int j = 0; j < r_struct[i].localNum; j++)
		{
			h_g[j + r_struct[i].base] = r_struct[i].h_local_g[j];
		}
	}
	while (k < h_kmax)
	{
		k++;
		if (k == 1)
		{
#pragma omp parallel num_threads(deviceCount)
			{
				int i = omp_get_thread_num();
				hipSetDevice(i);
				hipMemcpy(r_struct[i].d_local_p, r_struct[i].d_local_g, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToDevice);
				hipStreamSynchronize(r_struct[i].stream);
			}
		}
		else
		{
			if (k == 2)
			{
				h_phi_m = 0;

#pragma omp parallel num_threads(deviceCount)
				{
					int i = omp_get_thread_num();
					hipSetDevice(i);
					hipMemcpyAsync(r_struct[i].h_local_m_real, r_struct[i].d_local_m_real, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
					hipStreamSynchronize(r_struct[i].stream);
				}
				for (int i = 0; i < deviceCount; i++)
				{
					for (int j = 0; j < r_struct[i].localNum; j++)
					{
						h_phi_m += (r_struct[i].h_local_m_real[j] * r_struct[i].h_local_m_real[j]) / (r_struct[i].h_local_m_real[j] * r_struct[i].h_local_m_real[j] + h_sigma * h_sigma);
					}
				}
				h_lambda = h_d_square / h_phi_m;
			}
			else
			{
				h_lambda = h_lambda / 2;
			}

#pragma omp parallel num_threads(deviceCount)
			{
				int i = omp_get_thread_num();
				hipSetDevice(i);

				update_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_p0, r_struct[i].d_local_g0, r_struct[i].d_local_W,
					r_struct[i].d_local_m_temp, r_struct[i].d_local_p, r_struct[i].d_local_g, r_struct[i].d_local_m_real, r_struct[i].d_local_Wm, r_struct[i].d_local_Wv, r_struct[i].localNum);

				A_mult_v_col_sln << <r_struct[i].nBlocks, nThreadPerBlock, nThreadPerBlock * sizeof(double), r_struct[i].stream>> > (r_struct[i].d_local_d_fit_temp1, v_struct[i].d_Vz_mat_mc, r_struct[i].d_local_W,
					r_struct[i].d_local_m_temp, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx, nThreadPerBlock);
				A_mult_v_sum_sln << <r_struct[i].nBlocks_Am, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_d_fit_temp2, r_struct[i].d_local_d_fit_temp1, r_struct[i].d_local_m_temp,
					h_lambda, r_struct[i].localNum, r_struct[i].base, h_point_count, h_prism_count, r_struct[i].nBlocks);

				hipMemcpyAsync(r_struct[i].h_local_d_fit_temp, r_struct[i].d_local_d_fit_temp2, (h_prism_count + h_point_count) * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
				hipStreamSynchronize(r_struct[i].stream);
			}
			memset(h_d_fit_temp, 0, (h_prism_count + h_point_count) * sizeof(double));
			for (int i = 0; i < deviceCount; i++)
			{
				for (int j = 0; j < (h_prism_count + h_point_count); j++)
				{
					h_d_fit_temp[j] += r_struct[i].h_local_d_fit_temp[j];
				}
			}

#pragma omp parallel num_threads(deviceCount)
			{
				int i = omp_get_thread_num();
				hipSetDevice(i);
				hipMemcpyAsync(r_struct[i].d_d_fit_temp, h_d_fit_temp, (h_prism_count + h_point_count) * sizeof(double), hipMemcpyHostToDevice, r_struct[i].stream);
				g_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_g, v_struct[i].d_Vz_mat_mc, r_struct[i].d_local_W,
					i_struct[i].d_Vz, r_struct[i].d_d_fit_temp, h_lambda, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx);
				hipMemcpyAsync(r_struct[i].h_local_g, r_struct[i].d_local_g, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
				hipMemcpyAsync(r_struct[i].h_local_g0, r_struct[i].d_local_g0, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
				hipStreamSynchronize(r_struct[i].stream);
			}

#pragma omp parallel num_threads(deviceCount)
			{
				int i = omp_get_thread_num();
				hipSetDevice(i);
				for (int j = 0; j < r_struct[i].localNum; j++)
				{
					h_g[j + r_struct[i].base] = r_struct[i].h_local_g[j];
					h_g0[j + r_struct[i].base] = r_struct[i].h_local_g0[j];
				}
			}
			beta = beta_sln(h_g, h_g0, h_prism_count);
			//beta = vector_dot_product(h_g, h_g, h_prism_count) / vector_dot_product(h_g0, h_g0, h_prism_count);

#pragma omp parallel num_threads(deviceCount)
			{
				int i = omp_get_thread_num();
				hipSetDevice(i);
				p_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_p, r_struct[i].d_local_g, r_struct[i].d_local_p0, beta, r_struct[i].localNum);
				hipStreamSynchronize(r_struct[i].stream);
			}
		}

#pragma omp parallel num_threads(deviceCount)
		{
			int i = omp_get_thread_num();
			hipSetDevice(i);
			A_mult_v_col_sln << <r_struct[i].nBlocks, nThreadPerBlock,nThreadPerBlock * sizeof(double), r_struct[i].stream >> > (r_struct[i].d_local_q1, v_struct[i].d_Vz_mat_mc, r_struct[i].d_local_W, r_struct[i].d_local_p, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx, nThreadPerBlock);
			A_mult_v_sum_sln << <r_struct[i].nBlocks_Am, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_q2, r_struct[i].d_local_q1, r_struct[i].d_local_p, h_lambda, r_struct[i].localNum, r_struct[i].base, h_point_count, h_prism_count, r_struct[i].nBlocks);

			hipMemcpyAsync(r_struct[i].h_local_q, r_struct[i].d_local_q2, (h_prism_count + h_point_count) * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
			hipStreamSynchronize(r_struct[i].stream);
		}
		memset(h_q, 0, (h_prism_count + h_point_count) * sizeof(double));
		for (int i = 0; i < deviceCount; i++)
		{
			for (int j = 0; j < (h_prism_count + h_point_count); j++)
			{
				h_q[j] += r_struct[i].h_local_q[j];
			}
		}
		alpha = vector_dot_product(h_g, h_g, h_prism_count) / vector_dot_product(h_q, h_q, (h_prism_count + h_point_count));

#pragma omp parallel num_threads(deviceCount)
		{
			int i = omp_get_thread_num();
			hipSetDevice(i);
			m_sln << <r_struct[i].nBlocks, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_m_temp, r_struct[i].d_local_m_real,
				r_struct[i].d_local_p, r_struct[i].d_local_W, alpha, h_m_min, h_m_max, r_struct[i].localNum);
			G_mult_m_col_sln << <r_struct[i].nBlocks, nThreadPerBlock, nThreadPerBlock * sizeof(double), r_struct[i].stream >> > (r_struct[i].d_local_d_fit1, v_struct[i].d_Vz_mat_mc, r_struct[i].d_local_m_real, r_struct[i].localNum, r_struct[i].base, h_point_count, h_lx, nThreadPerBlock);
			G_mult_m_sum_sln << <r_struct[i].nBlocks_Gm, nThreadPerBlock, 0, r_struct[i].stream >> > (r_struct[i].d_local_d_fit2, r_struct[i].d_local_d_fit1, r_struct[i].localNum, h_point_count, r_struct[i].nBlocks);

			hipMemcpyAsync(r_struct[i].h_local_d_fit, r_struct[i].d_local_d_fit2, h_point_count * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
			hipStreamSynchronize(r_struct[i].stream);
		}
		memset(h_data_fitting, 0, h_point_count * sizeof(double));
		for (int i = 0; i < deviceCount; i++)
		{
			for (int j = 0; j < h_point_count; j++)
			{
				h_data_fitting[j] += r_struct[i].h_local_d_fit[j];
			}
		}
		for (int j = 0; j < h_point_count; j++)
		{
			h_data_misfit[j] = VzX.Vz[j] - h_data_fitting[j];
		}
		h_d_square = vector_dot_product(h_data_misfit, h_data_misfit, h_point_count);
		rms = sqrt(h_d_square / h_point_count);
		if (rms <= h_epsilon)
		{
			break;
		}
	}

#pragma omp parallel num_threads(deviceCount)
	{
		int i = omp_get_thread_num();
		hipSetDevice(i);
		hipMemcpyAsync(r_struct[i].h_local_m_real, r_struct[i].d_local_m_real, r_struct[i].localNum * sizeof(double), hipMemcpyDeviceToHost, r_struct[i].stream);
		hipStreamSynchronize(r_struct[i].stream);
		for (int j = 0; j < r_struct[i].localNum; j++)
		{
			inv_result[j + r_struct[i].base] = r_struct[i].h_local_m_real[j];
		}
		hipHostFree(i_struct[i].h_Vz);
		hipHostFree(i_struct[i].h_x_obs);
		hipHostFree(i_struct[i].h_y_obs);
		hipHostFree(i_struct[i].h_m1_x);
		hipHostFree(i_struct[i].h_m1_y);
		hipHostFree(i_struct[i].h_m1_z);
		hipHostFree(r_struct[i].h_local_g);
		hipHostFree(r_struct[i].h_local_g0);
		hipHostFree(r_struct[i].h_local_q);
		hipHostFree(r_struct[i].h_local_m_real);
		hipHostFree(r_struct[i].h_local_d_fit);
		hipHostFree(r_struct[i].h_local_d_fit_temp);
		hipFree(i_struct[i].d_Vz);
		hipFree(i_struct[i].d_x_obs);
		hipFree(i_struct[i].d_y_obs);
		hipFree(i_struct[i].d_m1_x);
		hipFree(i_struct[i].d_m1_y);
		hipFree(i_struct[i].d_m1_z);
		hipFree(v_struct[i].d_Vz_mat_mc);
		hipFree(r_struct[i].d_local_Wm);
		hipFree(r_struct[i].d_local_Wv);
		hipFree(r_struct[i].d_local_W);
		hipFree(r_struct[i].d_local_g);
		hipFree(r_struct[i].d_local_g0);
		hipFree(r_struct[i].d_local_p);
		hipFree(r_struct[i].d_local_p0);
		hipFree(r_struct[i].d_local_q1);
		hipFree(r_struct[i].d_local_q2);
		hipFree(r_struct[i].d_local_m_temp);
		hipFree(r_struct[i].d_local_m_real);
		hipFree(r_struct[i].d_local_d_fit1);
		hipFree(r_struct[i].d_local_d_fit2);
		hipFree(r_struct[i].d_local_d_fit_temp1);
		hipFree(r_struct[i].d_local_d_fit_temp2);
		hipFree(r_struct[i].d_d_fit_temp);
		hipStreamDestroy(r_struct[i].stream);
	}
	//free(h_data_misfit); free(h_data_fitting); free(h_g); free(h_g0); free(h_q); free(h_d_fit_temp);

	return inv_result;
}

__global__ void Vz_mat_mc_sln(double* Vz_mat_mc, double* x_obs, double* y_obs, double* m1_x, double* m1_y, double* m1_z, double z_obs, int lx, int lz, int point_count, int prism_count)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int obs_x, obs_y, prm_z;
	double r0, r1, r2, r3, r4, r5, r6, r7;
	double xt1, xt2, yt1, yt2, zt1, zt2;
	double d_G = 66.7;

	if (i < prism_count)
	{
		obs_x = (i % point_count) % lx;
		obs_y = (i % point_count) / lx;
		prm_z = i / point_count;
		xt1 = x_obs[obs_x] - m1_x[0]; xt2 = x_obs[obs_x] - m1_x[1];
		yt1 = y_obs[obs_y] - m1_y[0]; yt2 = y_obs[obs_y] - m1_y[1];
		zt1 = z_obs - m1_z[prm_z]; zt2 = z_obs - m1_z[prm_z + lz];

		r0 = sqrt(pow(xt1, 2) + pow(yt1, 2) + pow(zt1, 2));
		r1 = sqrt(pow(xt1, 2) + pow(yt1, 2) + pow(zt2, 2));
		r2 = sqrt(pow(xt1, 2) + pow(yt2, 2) + pow(zt1, 2));
		r3 = sqrt(pow(xt1, 2) + pow(yt2, 2) + pow(zt2, 2));
		r4 = sqrt(pow(xt2, 2) + pow(yt1, 2) + pow(zt1, 2));
		r5 = sqrt(pow(xt2, 2) + pow(yt1, 2) + pow(zt2, 2));
		r6 = sqrt(pow(xt2, 2) + pow(yt2, 2) + pow(zt1, 2));
		r7 = sqrt(pow(xt2, 2) + pow(yt2, 2) + pow(zt2, 2));
		Vz_mat_mc[i] += d_G * (-atan(xt1 * yt1 / zt1 / r0) + atan(xt1 * yt1 / zt2 / r1) + atan(xt1 * yt2 / zt1 / r2) - atan(xt1 * yt2 / zt2 / r3)
			+ atan(xt2 * yt1 / zt1 / r4) - atan(xt2 * yt1 / zt2 / r5) - atan(xt2 * yt2 / zt1 / r6) + atan(xt2 * yt2 / zt2 / r7));
	}
}

__global__ void W_init_sln(double* Wm, double* Wv, double* W, double* m_temp, double* m_real, double* Vz_mat_mc, double sigma, int localNum, int base, int point_count, int lx,double wn)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int i_base, m, n, px, py, pi, pj, pk, index;

	if (i < localNum)
	{
		Wm[i] = 0; m_temp[i] = 0; m_real[i] = 0;
		i_base = i + base;
		px = (i_base % point_count) % lx + 1;
		py = (i_base % point_count) / lx + 1;
		pk = i_base / point_count + 1;
		for (int j = 0; j < point_count; j++)
		{
			m = j % lx + 1; n = j / lx + 1;
			pi = abs(m - px) + 1;
			pj = abs(n - py) + 1;
			index = (pk - 1) * point_count + (pj - 1) * lx + pi - 1;
			Wm[i] += pow(Vz_mat_mc[index], 2);
		}
		/*Wm[i] = 1 / sqrt(sqrt(Wm[i]));
		Wv[i] = pow(sigma, 2) * Wm[i];*/
		Wm[i] = 1 / pow(Wm[i], wn);
		Wv[i] = pow(sigma, 2) * Wm[i];
		W[i] = sqrt(Wv[i]);
	}
}

__global__ void g0_sln(double* g, double* Vz_mat_mc, double* W, double* Vz, int localNum, int base, int point_count, int lx)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int i_base, m, n, px, py, pi, pj, pk, index;

	if (i < localNum)
	{
		g[i] = 0;
		i_base = i + base;
		px = (i_base % point_count) % lx + 1;
		py = (i_base % point_count) / lx + 1;
		pk = i_base / point_count + 1;
		for (int j = 0; j < point_count; j++)
		{
			m = j % lx + 1; n = j / lx + 1;
			pi = abs(m - px) + 1;
			pj = abs(n - py) + 1;
			index = (pk - 1) * point_count + (pj - 1) * lx + pi - 1;
			g[i] += Vz_mat_mc[index] * W[i] * Vz[j];
		}
	}
}

__global__ void A_mult_v_col_sln(double* q, double* Vz_mat_mc, double* W, double* vector, int localNum, int base, int point_count, int lx, int nThreadPerBlock)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int i_base, m, n, px, py, pi, pj, pk, index;
	double temp;
	extern __shared__ double V_temp_shared[];
	//hipMalloc((void**)V_temp_shared, nThreadPerBlock * sizeof(double));


	if (i < localNum)
	{
		i_base = i + base;
		px = (i_base % point_count) % lx + 1;
		py = (i_base % point_count) / lx + 1;
		pk = i_base / point_count + 1;
		for (int j = 0; j < point_count; j++)
		{
			m = j % lx + 1; n = j / lx + 1;
			pi = abs(m - px) + 1;
			pj = abs(n - py) + 1;
			index = (pk - 1) * point_count + (pj - 1) * lx + pi - 1;
			V_temp_shared[threadIdx.x] = Vz_mat_mc[index] * W[i] * vector[i];
			//*(V_temp_shared + threadIdx.x) = Vz_mat_mc[index] * W[i] * vector[i];
			__syncthreads();
			temp = 0;
			if (threadIdx.x == 0)
			{
				for (int k = 0; k < nThreadPerBlock; k++)
				{
					temp += V_temp_shared[k];
					//temp += *(V_temp_shared + k);
				}
				q[blockIdx.x * point_count + j] = temp;
			}
			__syncthreads();
		}
	}
}

__global__ void A_mult_v_sum_sln(double* q2, double* q1, double* vector, double lambda, int localNum, int base, int point_count, int prism_count, int nBlocks)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < (prism_count + point_count))
	{
		q2[i] = 0;
		if (i < point_count)
		{
			for (int j = 0; j < nBlocks; j++)
			{
				q2[i] += q1[j * point_count + i];
			}
		}
		else if (i >= (point_count + base) && i < (point_count + base + localNum))
		{
			q2[i] = sqrt(lambda) * vector[i - point_count - base];
		}
	}
}

__global__ void m_sln(double* m_temp, double* m_real, double* p, double* W, double alpha, double m_min, double m_max, int localNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < localNum)
	{
		m_temp[i] += alpha * p[i];
		m_real[i] = W[i] * m_temp[i];
		if (m_real[i] < m_min)
		{
			m_real[i] = m_min;
		}
		else if (m_real[i] > m_max)
		{
			m_real[i] = m_max;
		}
	}
}

__global__ void G_mult_m_col_sln(double* d_fit, double* Vz_mat_mc, double* m_temp, int localNum, int base, int point_count, int lx, int nThreadPerBlock)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int i_base, m, n, px, py, pi, pj, pk, index;
	double temp;
	extern __shared__ double V_temp_shared[];
	//hipMalloc((void**)V_temp_shared, nThreadPerBlock * sizeof(double));

	if (i < localNum)
	{
		i_base = i + base;
		px = (i_base % point_count) % lx + 1;
		py = (i_base % point_count) / lx + 1;
		pk = i_base / point_count + 1;
		for (int j = 0; j < point_count; j++)
		{
			m = j % lx + 1; n = j / lx + 1;
			pi = abs(m - px) + 1;
			pj = abs(n - py) + 1;
			index = (pk - 1) * point_count + (pj - 1) * lx + pi - 1;
			V_temp_shared[threadIdx.x] = Vz_mat_mc[index] * m_temp[i];
			//*(V_temp_shared + threadIdx.x) = Vz_mat_mc[index] * m_temp[i];
			__syncthreads();
			temp = 0;
			if (threadIdx.x == 0)
			{
				for (int k = 0; k < nThreadPerBlock; k++)
				{
					temp += V_temp_shared[k];
					//temp += *(V_temp_shared + k);
				}
				d_fit[blockIdx.x * point_count + j] = temp;
			}
			__syncthreads();
		}
	}
}

__global__ void G_mult_m_sum_sln(double* d_fit2, double* d_fit1, int localNum, int point_count, int nBlocks)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < point_count)
	{
		d_fit2[i] = 0;
		for (int j = 0; j < nBlocks; j++)
		{
			d_fit2[i] += d_fit1[j * point_count + i];
		}
	}
}

__global__ void p_sln(double* p, double* g, double* p0, double beta, int localNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < localNum)
	{
		p[i] = g[i] + beta * p0[i];
	}
}

__global__ void update_sln(double* p0, double* g0, double* W, double* m_temp, double* p, double* g, double* m_real, double* Wm, double* Wv, int localNum)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < localNum)
	{
		p0[i] = p[i];
		g0[i] = g[i];
		W[i] = sqrt(pow(m_real[i], 2) * Wm[i] + Wv[i]);
		m_temp[i] = m_real[i] / W[i];
	}
}

__global__ void g_sln(double* g, double* Vz_mat_mc, double* W, double* Vz, double* d_fit_temp, double lambda, int localNum, int base, int point_count, int lx)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int i_base, m, n, px, py, pi, pj, pk, index;

	if (i < localNum)
	{
		g[i] = 0;
		i_base = i + base;
		px = (i_base % point_count) % lx + 1;
		py = (i_base % point_count) / lx + 1;
		pk = i_base / point_count + 1;
		for (int j = 0; j < point_count; j++)
		{
			m = j % lx + 1; n = j / lx + 1;
			pi = abs(m - px) + 1;
			pj = abs(n - py) + 1;
			index = (pk - 1) * point_count + (pj - 1) * lx + pi - 1;
			g[i] += Vz_mat_mc[index] * W[i] * (Vz[j] - d_fit_temp[j]);
		}
		g[i] += sqrt(lambda) * (0 - d_fit_temp[i_base + point_count]);
	}
}

double beta_sln(double* g, double* g0, int prism_count)
{
	double beta;
	double beta_temp = 0;
	for (int i = 0; i < prism_count; i++)
	{
		beta_temp += g[i] * (g[i] - g0[i]);
	}
	beta = beta_temp / vector_dot_product(g0, g0, prism_count);
	return beta;
}

double vector_dot_product(double* a, double* b, int count)
{
	double result = 0;
	for (int vi = 0; vi < count; vi++)
	{
		result += a[vi] * b[vi];
	}
	return result;
}
